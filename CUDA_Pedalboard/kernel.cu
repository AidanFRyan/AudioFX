#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "RtAudio.h"
#include <cmath>
#include <iostream>
#include <list>

#define TWELFTH 1.0594630943592952646

#include <iostream>

using namespace std;

__global__ void cudaDelay(double* a, double* prev, unsigned int numFrames, int numChannels, unsigned int d, double b) {
	unsigned int offset = gridDim.x * blockDim.x;
	for (unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; i < numFrames*numChannels; i += offset) {
		//unsigned int t = i * numChannels;
		//printf("%d %f\n", i, a[i]);
		if (prev != 0 && i < d*numChannels) {
			a[i] += b*prev[numFrames*numChannels - i];
		}
		else if(prev != 0) {
			a[i] += b * a[i - d*numChannels];
		}
		//printf("%d\t%f\n", a[t + j]);
	}
}

__global__ void monoOneToStereo(double* in, double* a, unsigned int numFrames, int numChannels) {
	unsigned int offset = gridDim.x * blockDim.x;
	for (unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; i < numFrames; i += offset) {
		unsigned int t = i * numChannels;
		for (int j = 0; j < numChannels; ++j) {
			a[t + j] = in[t];
			//printf("%f\n", a[t + j]);
		}
	}
}

class Pedal {
public:
	virtual void process(double* audio, int n, int c) = 0;
	void enable() {
		enabled = true;
	}
	void disable() {
		enabled = false;
	}
	bool enabled;
	list<double*> prev;
};

class Delay : public Pedal {
public:
	Delay() {
		//prev = 0;
		delay = 0;
		enabled = true;
	}
	virtual void process(double* a, int n, int c) {
		if (!enabled)
			return;
		cudaDelay << <1, 512 >> > (a, prev.back(), n, c, delay, blend);
		hipDeviceSynchronize();
		if (prev.size() == 0) {
			double* t;
			hipMalloc((void**)& t, sizeof(double) * n * c);
			prev.push_back(t);
		}
		hipMemcpy(prev.back(), a, sizeof(double) * n * c, hipMemcpyDeviceToDevice);

	}
	void setDelay(int samples) {
		delay = samples;
	}
	void setBlend(double b) {
		blend = b;
	}
	int delay;
	double blend;
};

class Pedalboard {
public:
	Pedalboard() {
		audio = 0;
		dev = 0;
	}

	void allOff() {
	}

	void allOn() {
	}

	void processEffects(double* dev, int numFrames, int numChannels) {
		for(list<Pedal*>::iterator i = effects.begin(); i != effects.end(); i++){
			(*i)->process(dev, numFrames, numChannels);
		}		
	}

	double *audio, *dev;
	list<Pedal*> effects;
};

int record(void* outputBuffer, void* inputBuffer, unsigned int nBufferFrames, double streamTime, RtAudioStreamStatus status, void* userData) {
	if (status) {
		printf("Audio error\n");
		return -1;
	}

	double *in = (double*)inputBuffer, *out = (double*)outputBuffer, *d_in, *d_out;
	Pedalboard* p = (Pedalboard*)userData;
	

	hipMalloc((void**)& d_in, sizeof(double) * nBufferFrames * 2);
	hipMalloc((void**)& d_out, sizeof(double) * nBufferFrames * 2);

	hipMemcpy(d_in, in, sizeof(double) * nBufferFrames * 2, hipMemcpyHostToDevice);
	monoOneToStereo << <1, 512 >> > (d_in, d_out, nBufferFrames, 2);
	hipDeviceSynchronize();

	hipFree(d_in);
	p->processEffects(d_out, nBufferFrames, 2);

	hipMemcpy(out, d_out, sizeof(double) * nBufferFrames * 2, hipMemcpyDeviceToHost);
	hipFree(d_out);

	return 0;
}

int main() {
	int count;
	hipGetDeviceCount(&count);
	printf("CUDA Devices: %d\n", count);
	RtAudio audio;
	RtAudio::StreamParameters inParam, outParam;
	RtAudio::DeviceInfo info;

	Pedalboard pedal;
	Delay* d = new Delay();
	pedal.effects.push_back(d);
	d->setDelay(120);
	d->setBlend(1);
	d->disable();

	for (int i = 0; i < audio.getDeviceCount(); ++i) {
		info = audio.getDeviceInfo(i);
		printf("Device %d: name: %s inChannels: %d outChannels: %d pref: %d\n", i, info.name.c_str(), info.inputChannels, info.outputChannels, info.preferredSampleRate);
	}

	int inS, outS;
	//cout << "Enter input device: ";
	//cin >> inS;
	//cout << "Enter output device: ";
	//cin >> outS;
	inS = 8;
	outS = 6;

	inParam.deviceId = inS;
	outParam.deviceId = outS;
	inParam.nChannels = audio.getDeviceInfo(inS).inputChannels;
	outParam.nChannels = audio.getDeviceInfo(outS).outputChannels;
	inParam.firstChannel = 0;
	outParam.firstChannel = 0;
	RtAudio::StreamOptions options;
	options.flags = RTAUDIO_MINIMIZE_LATENCY;
	unsigned int sampleRate = audio.getDeviceInfo(inS).preferredSampleRate, bufferFrames = 8192;
	audio.openStream(&outParam, &inParam, RTAUDIO_FLOAT64, sampleRate, &bufferFrames, &record, &pedal, &options);
	audio.startStream();

	cout << "\nRecording ... press enter to quit\n";

	string s;
	cin >> s;

	audio.stopStream();

	delete d;
	return 0;
}